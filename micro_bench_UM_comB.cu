/* 
Author: Janki Bhimani
Northeastern University
Email: bhimanijanki@gmail.com
 */



#include <hip/hip_runtime.h>
#include <stdio.h>
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(float *a, int offset, int x)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int p;
  int t = offset + ((i*(x))+j);
{
  float q = (float)t;
  float s = sinf(q); 
  float c = cosf(q);
  a[t] = a[t] + sqrtf(s*s+c*c); //adding 1 to a
  for(p=0;p<1;p++)
  {
	q = sinf(q); 
  	q = cosf(q);
	q = sqrtf(s*s+c*c);
  }
}
}

float maxError(float *a, int n) 
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int blockSize = 1024, nStreams = sqrt(atoi(argv[2]));
  int x = atoi(argv[1]);
  const int n = x *x * blockSize ;
  const int streamSize = n / nStreams/ nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);
   
  int devId = 0;
  if (argc > 3) devId = atoi(argv[3]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );
  dim3 block(32, 32);
  dim3 grid((sqrt(n))/32,(sqrt(n))/32); 
  dim3 grid1((sqrt(n))/nStreams/32, (sqrt(n))/nStreams/32);
  x= x* nStreams;

  float ms, msk, seq, aloc; // elapsed time in milliseconds
  
  // create events and streams
  hipEvent_t startaloc, stopaloc, startEvent, stopEvent, startKernel, stopKernel, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startaloc) );
  checkCuda( hipEventCreate(&stopaloc) );
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&startKernel) );
  checkCuda( hipEventCreate(&stopKernel) );
  checkCuda( hipEventCreate(&dummyEvent) );
for (int i = 0; i < nStreams* nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );
   checkCuda( hipEventRecord(startaloc,0) ); 
  float *a = (float*)malloc(bytes) ;     
  checkCuda( hipMallocManaged((void**)&a, bytes) ); // device
  checkCuda( hipEventRecord(stopaloc, 0) );
  checkCuda( hipEventSynchronize(stopaloc) );
  checkCuda( hipEventElapsedTime(&aloc, startaloc, stopaloc) );
  // baseline case - sequential transfer and execute
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  checkCuda( hipEventRecord(startKernel,0) );
  kernel<<<grid,block>>>(a, 0, sqrt(n));
    checkCuda(hipDeviceSynchronize());
  checkCuda( hipEventRecord(stopKernel, 0) );
  checkCuda( hipEventSynchronize(stopKernel) );
  checkCuda( hipEventElapsedTime(&msk, startKernel, stopKernel) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
seq = ms;  
printf("Time for seq transfer and execute (ms): %f\n", ms+aloc);
printf("Time for kernel execute (ms): %f\n", msk);
printf("Bytes for sequential transfer (bytes): %d\n", bytes);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  for (int i = 0; i < nStreams* nStreams; ++i) {
    int offset = i * streamSize;
    kernel<<<grid1, block, 0, stream[i]>>>(a, offset, sqrt(n)/nStreams);
    checkCuda(hipDeviceSynchronize());
  }
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Scheduling scheme type I transfer and execute (ms): %f\n", ms+aloc);
  printf("  max error: %e\n", maxError(a, n));

  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );

  for (int i = 0; i < nStreams* nStreams; ++i)
  {
    int offset = i * streamSize;
    kernel<<<grid1, block, 0, stream[i]>>>(a, offset, sqrt(n)/nStreams);
    checkCuda(hipDeviceSynchronize());
  }
 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Scheduling scheme type II transfer and execute (ms): %f\n", ms+aloc);
  printf("  max error: %e\n", maxError(a, n));
  printf("% Overlap (%): %f\n", (seq-ms)/seq*100);
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams* nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(a);
  //cudaFree(a);

  return 0;
}

